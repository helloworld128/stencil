#include "hip/hip_runtime.h"
#include "common.h"
#include <stdio.h>
#define XBS 32
#define YBS 4
#define ZBS 4
const int XX = XBS + 2;
const int YY = YBS + 2;
const int ZZ = ZBS + 2;
const char* version_name = "A naive base-line";

void create_dist_grid(dist_grid_info_t *grid_info, int stencil_type) {
    grid_info->halo_size_x = 1;
    grid_info->halo_size_y = 1;
    grid_info->halo_size_z = 1;
}

void destroy_dist_grid(dist_grid_info_t *grid_info) {

}

__global__ void stencil_7_naive_kernel_1step(cptr_t in, ptr_t out, \
                                int nx, int ny, int nz, \
                                int halo_x, int halo_y, int halo_z) {
    int tx = threadIdx.x + blockDim.x * blockIdx.x;
    int ty = threadIdx.y + blockDim.y * blockIdx.y;
    int tz = threadIdx.z + blockDim.z * blockIdx.z;
    if(tx < nx && ty < ny && tz < nz) {
        int ldx = nx + halo_x * 2;
        int ldy = ny + halo_y * 2;
        int x = tx + halo_x;
        int y = ty + halo_y;
        int z = tz + halo_z;

        __shared__ double s_in[XX * YY * ZZ];

        int xx = threadIdx.x + 1;
        int yy = threadIdx.y + 1;
        int zz = threadIdx.z + 1;

        int base1 = xx + XX * (yy + YY * zz);
        int base2 = x + ldx * (y + ldy * z);
        int bz1 = XX * YY;
        int bz2 = ldx * ldy;

        s_in[base1] = in[base2];
        int dx = -(xx == 1) + (xx == XBS);
        int dy = -(yy == 1) + (yy == YBS);
        int dz = -(zz == 1) + (zz == ZBS);
        if(dx) s_in[base1 + dx] = in[base2 + dx];
        if(dy) s_in[base1 + XX * dy] = in[base2 + ldx * dy];
        if(dz) s_in[base1 + bz1 * dz] = in[base2 + bz2 * dz];

        __syncthreads();

        out[base2] \
            = ALPHA_ZZZ * s_in[base1] \
            + ALPHA_NZZ * s_in[base1 - 1] \
            + ALPHA_PZZ * s_in[base1 + 1] \
            + ALPHA_ZNZ * s_in[base1 - XX] \
            + ALPHA_ZPZ * s_in[base1 + XX] \
            + ALPHA_ZZN * s_in[base1 - bz1] \
            + ALPHA_ZZP * s_in[base1 + bz1];
    }
}

__global__ void stencil_27_naive_kernel_1step(cptr_t in, ptr_t out, \
                                int nx, int ny, int nz, \
                                int halo_x, int halo_y, int halo_z) {
    int tx = threadIdx.x + blockDim.x * blockIdx.x;
    int ty = threadIdx.y + blockDim.y * blockIdx.y;
    int tz = threadIdx.z + blockDim.z * blockIdx.z;
    if(tx < nx && ty < ny && tz < nz) {
        int ldx = nx + halo_x * 2;
        int ldy = ny + halo_y * 2;
        int x = tx + halo_x;
        int y = ty + halo_y;
        int z = tz + halo_z;

        __shared__ double s_in[ZZ * YY * XX];

        int xx = threadIdx.x + 1;
        int yy = threadIdx.y + 1;
        int zz = threadIdx.z + 1;

        int dx = -(xx == 1) + (xx == XBS);
        int dy = -(yy == 1) + (yy == YBS);
        int dz = -(zz == 1) + (zz == ZBS);

        int base1 = xx + XX * (yy + YY * zz);
        int base2 = x + ldx * (y + ldy * z);
        int bz1 = XX * YY;
        int bz2 = ldx * ldy;

        s_in[base1] = in[base2];
        if(dx) s_in[base1 + dx] = in[base2 + dx];
        if(dy) s_in[base1 + XX * dy] = in[base2 + ldx * dy];
        if(dz) s_in[base1 + bz1 * dz] = in[base2 + bz2 * dz];
        if(dx && dy) s_in[base1 + XX * dy + dx] = in[base2 + ldx * dy + dx];
        if(dx && dz) s_in[base1 + bz1 * dz + dx] = in[base2 + bz2 * dz + dx];
        if(dy && dz) s_in[base1 + bz1 * dz + XX * dy] = in[base2 + bz2 * dz + ldx * dy];
        if(dx && dy && dz) s_in[base1 + bz1 * dz + XX * dy + dx] = in[base2 + bz2 * dz + ldx * dy + dx];
        // int minx = (dx < 0) ? dx : 0;
        // int maxx = (dx < 0) ? 0 : dx;
        // int miny = (dy < 0) ? dy : 0;
        // int maxy = (dy < 0) ? 0 : dy;
        // int minz = (dz < 0) ? dz : 0;
        // int maxz = (dz < 0) ? 0 : dz;
        // for(int i = minz; i <= maxz; i++){
        //     for(int j = miny; j <= maxy; j++){
        //         for(int k = minx; k <= maxx; k++){
        //             s_in[zz + i][yy + j][xx + k] = in[INDEX(x + k, y + j, z + i, ldx, ldy)];
        //         }
        //     }
        // }

        __syncthreads();

        out[INDEX(x, y, z, ldx, ldy)] \
            = ALPHA_ZZZ * s_in[base1] \
            + ALPHA_NZZ * s_in[base1 - 1] \
            + ALPHA_PZZ * s_in[base1 + 1] \
            + ALPHA_ZNZ * s_in[base1 - XX] \
            + ALPHA_ZPZ * s_in[base1 + XX] \
            + ALPHA_ZZN * s_in[base1 - bz1] \
            + ALPHA_ZZP * s_in[base1 + bz1] \
            + ALPHA_NNZ * s_in[base1 - XX - 1] \
            + ALPHA_PNZ * s_in[base1 - XX + 1] \
            + ALPHA_NPZ * s_in[base1 + XX - 1] \
            + ALPHA_PPZ * s_in[base1 + XX + 1] \
            + ALPHA_NZN * s_in[base1 - bz1 - 1] \
            + ALPHA_PZN * s_in[base1 - bz1 + 1] \
            + ALPHA_NZP * s_in[base1 + bz1 - 1] \
            + ALPHA_PZP * s_in[base1 + bz1 + 1] \
            + ALPHA_ZNN * s_in[base1 - bz1 - XX] \
            + ALPHA_ZPN * s_in[base1 - bz1 + XX] \
            + ALPHA_ZNP * s_in[base1 + bz1 - XX] \
            + ALPHA_ZPP * s_in[base1 + bz1 + XX] \
            + ALPHA_NNN * s_in[base1 - bz1 - XX - 1] \
            + ALPHA_PNN * s_in[base1 - bz1 - XX + 1] \
            + ALPHA_NPN * s_in[base1 - bz1 + XX - 1] \
            + ALPHA_PPN * s_in[base1 - bz1 + XX + 1] \
            + ALPHA_NNP * s_in[base1 + bz1 - XX - 1] \
            + ALPHA_PNP * s_in[base1 + bz1 - XX + 1] \
            + ALPHA_NPP * s_in[base1 + bz1 + XX - 1] \
            + ALPHA_PPP * s_in[base1 + bz1 + XX + 1];
    }
}

inline int ceiling(int num, int den) {
    return (num - 1) / den + 1;
}

ptr_t stencil_7(ptr_t grid, ptr_t aux, const dist_grid_info_t *grid_info, int nt) {
    ptr_t buffer[2] = {grid, aux};
    int nx = grid_info->global_size_x;
    int ny = grid_info->global_size_y;
    int nz = grid_info->global_size_z;
    // hipDeviceProp_t devp;
    // int dev = 0;
    // hipGetDeviceProperties(&devp, dev);
    // printf("%d\n",  devp.maxThreadsPerBlock);
    dim3 grid_size (ceiling(nx, XBS), ceiling(ny, YBS), ceiling(nz, ZBS));
    dim3 block_size (XBS, YBS, ZBS);
    for(int t = 0; t < nt; ++t) {
        stencil_7_naive_kernel_1step<<<grid_size, block_size>>>(\
            buffer[t % 2], buffer[(t + 1) % 2], nx, ny, nz, \
                grid_info->halo_size_x, grid_info->halo_size_y, grid_info->halo_size_z);
    }
    return buffer[nt % 2];
}

ptr_t stencil_27(ptr_t grid, ptr_t aux, const dist_grid_info_t *grid_info, int nt) {
    ptr_t buffer[2] = {grid, aux};
    int nx = grid_info->global_size_x;
    int ny = grid_info->global_size_y;
    int nz = grid_info->global_size_z;
    dim3 grid_size (ceiling(nx, XBS), ceiling(ny, YBS), ceiling(nz, ZBS));
    dim3 block_size (XBS, YBS, ZBS);
    for(int t = 0; t < nt; ++t) {
        stencil_27_naive_kernel_1step<<<grid_size, block_size>>>(\
            buffer[t % 2], buffer[(t + 1) % 2], nx, ny, nz, \
                grid_info->halo_size_x, grid_info->halo_size_y, grid_info->halo_size_z);
    }
    return buffer[nt % 2];
}